#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc,char **argv)
{
    printf("%s Starting...\n",argv[0]);

    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if(error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n->%s",
                        (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    
    if(deviceCount == 0)
    {
        printf("There are no available device(s) that support CUDA\n");
    }
    else
    {
        printf("Detected %d CUDA Capable device(s)\n",deviceCount);
    }

    int dev,driverVersion = 0,runtimeVersion = 0;

    dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,dev);
    printf("Device %d: \"%s\"\n",dev,deviceProp.name);
    printf("最大线程数/块：%d\n",deviceProp.maxThreadsPerBlock);
    printf("最大块维度：%d,%d,%d\n",deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
    printf("最大网格维度：%d,%d,%d\n",deviceProp.maxGridSize[0],deviceProp.maxGridSize[1],deviceProp.maxGridSize[2]);
    return 0;
    
    
}